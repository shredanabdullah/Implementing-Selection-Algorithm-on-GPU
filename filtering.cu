#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 1000000
#define THREADS_PER_BLOCK 256
#define TILE_SIZE 256 // tile size for the shared memory

// Predicate condition
__device__ bool predicate_condition(int element) {
    return (element % 2 != 0) && (element >= 50 && element <= 60);
}

__global__ void filter_kernel(const int* data, int* result_array, int size) {
    __shared__ int tile[TILE_SIZE]; // Declaring the shared memory tile
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Calculating global thread ID

    // Checking if thread is within bounds
    if (tid < size) {
        tile[threadIdx.x] = data[tid]; // Loading data into shared memory
        __syncthreads(); // Synchronizing threads within the block

        // Checking if element satisfies predicate condition
        if (predicate_condition(tile[threadIdx.x])) {
            result_array[tid] = tile[threadIdx.x]; // If true, write element to result array
        } else {
            result_array[tid] = -1; // If not, write -1 to result array
        }
    }
}

__global__ void compact_kernel(const int* result_array, int* final_output, int* count, int size) {
    __shared__ int tile[TILE_SIZE]; // Shared memory for the tile
    __shared__ int tile_count; // Shared memory for counting valid elements in the tile

    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Calculating global thread ID

    // Initialize shared memory count to 0
    if (threadIdx.x == 0) {
        tile_count = 0;
    }
    __syncthreads();

    // Load data into shared memory and check for valid elements
    if (tid < size) {
        if (result_array[tid] != -1) {
            int local_index = atomicAdd(&tile_count, 1);
            tile[local_index] = result_array[tid];
        }
    }
    __syncthreads();

    // Compact valid elements from shared memory to global memory
    if (threadIdx.x < tile_count) {
        int global_index = atomicAdd(count, 1);
        final_output[global_index] = tile[threadIdx.x];
    }
}

int main() {
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Allocate memory on the GPU for input data and result array
    int *data, *result_array, *final_output, *dev_count;
    hipMalloc((void**)&data, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&result_array, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&final_output, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&dev_count, sizeof(int));

    // Allocate memory on the host (CPU) for input data
    int* host_data = (int*)malloc(ARRAY_SIZE * sizeof(int));
    // Initialize input data with values from 1000000 to 1 (unsorted)
    for (int i = 0; i < ARRAY_SIZE; ++i) {
        host_data[i] = ARRAY_SIZE - i;
    }

    // Copy input data from host to device asynchronously using stream1
    hipMemcpyAsync(data, host_data, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice, stream1);

    // Create Thrust device pointer and sort data on the GPU using stream1
    thrust::device_ptr<int> dev_ptr(data);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream1);
    thrust::sort(thrust::cuda::par.on(stream1), dev_ptr, dev_ptr + ARRAY_SIZE);

    // Launch the filter kernel on the GPU using stream1
    filter_kernel<<<(ARRAY_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, 0, stream1>>>(data, result_array, ARRAY_SIZE);
    hipEventRecord(stop, stream1);
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Initialize device count to 0
    hipMemsetAsync(dev_count, 0, sizeof(int), stream2);

    // Launch the compact kernel on the GPU using stream2
    compact_kernel<<<(ARRAY_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, 0, stream2>>>(result_array, final_output, dev_count, ARRAY_SIZE);

    // Allocate memory on the host for the final output array and count
    int* host_final_output = (int*)malloc(ARRAY_SIZE * sizeof(int));
    int host_count;

    // Copy the final output array and count from device to host asynchronously using stream2
    hipMemcpyAsync(host_final_output, final_output, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(&host_count, dev_count, sizeof(int), hipMemcpyDeviceToHost, stream2);

    // Synchronize stream2 to ensure all operations are complete before accessing the results
    hipStreamSynchronize(stream2);

    FILE *output_file = fopen("output.txt", "w");

    // Write the elements satisfying the predicate condition to the output file
    fprintf(output_file, "Elements satisfying the predicate condition:\n");
    for (int i = 0; i < host_count; ++i) {
        fprintf(output_file, "%d\n", host_final_output[i]);
    }

    fprintf(output_file, "Time taken for analysis: %f milliseconds\n", milliseconds);

    fclose(output_file);

    
    hipFree(data);
    hipFree(result_array);
    hipFree(final_output);
    hipFree(dev_count);

    
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    free(host_data);
    free(host_final_output);

    return 0;
}
